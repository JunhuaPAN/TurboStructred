#include "hip/hip_runtime.h"
#include "CUDAManager.cu.h"
#include "hip/hip_runtime.h"
#include "device_functions.hpp"

//__global__ void CUDAManager::ComputeFluxesInitKernel()  {
//	//Create RiemannSolver class in GPU memory
//	(*_deviceRiemannSolver) = new RoeSolverPerfectGasEOS(_parameters.gamma, _parameters.eps, _parameters.opPressure);
//};
//
//__global__ void CUDAManager::CUDAInitComputeFluxes(CUDAComputeFluxesParameters parameters) {
//	//Compute optimal run parameters and hardware capability
//	hipError_t cudaStatus;
//
//	//Initialize the grid and block dimensions here
//	int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
//	int maxSharedMemory = deviceProp.sharedMemPerBlock;
//	int warpSize = deviceProp.warpSize;
//	int nFaces = parameters.nFaces;
//	int nVariables = parameters.nVariables;
//
//	//Determine shared memory threads limit
//	int deviceFaceMemory = 0;
//	deviceFaceMemory = 3 * nVariables * sizeof(double);
//	deviceFaceMemory += sizeof(Vector);
//	int maxThreadsMemory = 1 + (maxSharedMemory - 1) / deviceFaceMemory;
//
//	//Block dimensions
//	int nThreads = std::min(maxThreadsMemory, maxThreadsPerBlock);
//	DimBlock.x = nThreads;
//	DimBlock.y = 1;
//	DimBlock.z = 1;
//
//	//Grid dimensions
//	DimGrid.x = 1 + (nFaces - 1) / nThreads;
//	DimGrid.y = 1;
//	DimGrid.z = 1;
//		
//
//	//Fill device constant memory once
//	cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(&_parameters), &parameters, sizeof(CUDAComputeFluxesParameters));
//	if (cudaStatus != hipSuccess) {
//		std::cerr<<"hipDeviceSynchronize returned error code "<<cudaStatus<<" after launching matrixMultiplyShared!"<<std::endl;
//		CUDAFinalizeComputeFluxes(); exit(0);
//	};
//
//	//Create RiemannSolver class in GPU memory
//	hipMalloc(&_deviceRiemannSolver, sizeof(RoeSolverPerfectGasEOS*));
//
//	//Allocate device memory for data storage
//	int size = parameters.nVariables * nFaces;
//
//	cudaStatus = hipMalloc((void**)&deviceLeftStates, size * sizeof(double));
//	if (cudaStatus != hipSuccess) {
//		std::cerr<<"hipMalloc failed!"<<std::endl;
//		CUDAFinalizeComputeFluxes(); exit(0);
//	};
//
//	cudaStatus = hipMalloc((void**)&deviceRightStates, size * sizeof(double));
//	if (cudaStatus != hipSuccess) {
//		std::cerr<<"hipMalloc failed!"<<std::endl;
//		CUDAFinalizeComputeFluxes(); exit(0);
//	};
//
//	cudaStatus = hipMalloc((void**)&deviceFluxes, size * sizeof(double));
//	if (cudaStatus != hipSuccess) {
//		std::cerr<<"hipMalloc failed!"<<std::endl;
//		CUDAFinalizeComputeFluxes(); exit(0);
//	};
//
//	cudaStatus = hipMalloc((void**)&deviceFaceNormals, nFaces * sizeof(Vector));
//	if (cudaStatus != hipSuccess) {
//		std::cerr<<"hipMalloc failed!"<<std::endl;
//		CUDAFinalizeComputeFluxes(); exit(0);
//	};
//
//	//Make device side preparations
//	ComputeFluxesInitKernel<<<DimGrid, DimBlock>>>();
//};
//
//__global__ void CUDAManager::CUDAFinalizeComputeFluxes() {
//	//Clean device
//	ComputeFluxesFinalizeKernel<<<DimGrid, DimBlock>>>();
//
//	//Free device memory
//	hipFree(deviceLeftStates);
//	hipFree(deviceRightStates);
//	hipFree(deviceFluxes);
//	hipFree(deviceFaceNormals);
//	hipFree(_deviceRiemannSolver);
//};
//
//__global__ void CUDAManager::CUDAComputeFluxes(double* LeftStates, double *RightStates, Vector *faceNormals, double *Fluxes, double* Velocities) {
//	hipError_t cudaStatus;
//	int size = _parameters.nVariables * _parameters.nFaces;
//
//	//Copy left and right states to device memory
//	cudaStatus = hipMemcpy(deviceLeftStates, LeftStates, size * sizeof(double), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		std::cerr<<"hipMemcpy failed!"<<std::endl;
//		CUDAFinalizeComputeFluxes(); exit(0);
//	};
//
//	cudaStatus = hipMemcpy(deviceRightStates, RightStates, size * sizeof(double), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		std::cerr<<"hipMemcpy failed!"<<std::endl;
//		CUDAFinalizeComputeFluxes(); exit(0);
//	};
//
//	//Copy face normals to GPU buffer
//	cudaStatus = hipMemcpy(deviceRightStates, RightStates, size * sizeof(double), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		std::cerr<<"hipMemcpy failed!"<<std::endl;
//		CUDAFinalizeComputeFluxes(); exit(0);
//	};
//
//	//Launch calculation kernel
//	ComputeFluxesSharedKernel<<<DimGrid, DimBlock>>>(deviceLeftStates, deviceRightStates, deviceFaceNormals, deviceFluxes);
//
//	//Copy fluxes to host memory
//};


__host__ void CUDAManager::Init(CUDAConfiguration config) {
	int deviceCount;
	hipError_t cudaStatus = hipGetDeviceCount(&deviceCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipGetDeviceCount failed!");
		return;
	}	

	for (int dev = 0; dev < deviceCount; dev++) {
		hipGetDeviceProperties(&deviceProp, dev);

		if (dev == 0) {
			if (deviceProp.major == 9999 && deviceProp.minor == 9999) {
				std::cout<<"No CUDA GPU has been detected"<<std::endl;
				return;
			} else if (deviceCount == 1) {
				std::cout<<"There is 1 device supporting CUDA"<<std::endl;
			} else {
				std::cout<<"There are "<<deviceCount<<" devices supporting CUDA"<<std::endl;
			}
		}

		std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
		std::cout<<" Computational Capabilities: "<< deviceProp.major<< "."<< deviceProp.minor<<std::endl;
		std::cout<<" Maximum global memory size: "<< deviceProp.totalGlobalMem<<std::endl;
		std::cout<<" Maximum constant memory size: "<< deviceProp.totalConstMem<<std::endl;
		std::cout<<" Maximum shared memory size per block: "<< deviceProp.sharedMemPerBlock<<std::endl;
		std::cout<<" Maximum block dimensions: "<< deviceProp.maxThreadsDim[0]<< " x "<<
													deviceProp.maxThreadsDim[1]<< " x "<<
													deviceProp.maxThreadsDim[2]<<std::endl;
		std::cout<<" Maximum grid dimensions: "<< deviceProp.maxGridSize[0]<< " x "<<
													deviceProp.maxGridSize[1]<< " x "<<
													deviceProp.maxGridSize[2]<<std::endl;
		std::cout<<" Warp size: "<<deviceProp.warpSize<<std::endl;
	};

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);	
	if (cudaStatus != hipSuccess) {
		std::cerr<<"hipSetDevice failed!  Do you have a CUDA-capable GPU installed?";
		exit(0);
	};

	// Get device properties
	hipGetDeviceProperties(&deviceProp, 0);
};